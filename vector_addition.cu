
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

// vector addition on device
template<typename Scalar_type, typename Int_type>
__global__ void vector_addition(const Scalar_type* A, const Scalar_type* B,
                                Scalar_type* C, const Int_type N){

  int global_thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;

  if(global_thread_id < N){
    C[global_thread_id] = A[global_thread_id] + B[global_thread_id];
  }
}

// check results on host
template<typename Vector_type>
void check(Vector_type& A, Vector_type& B, Vector_type& C){
  for(int i = 0; i < A.size(); ++i){
    assert(C[i] == A[i] + B[i]);
  }
}

int main(){
  constexpr int N = 1 << 16;
  constexpr size_t bytes = sizeof(int)*N;

  // host memory
  std::vector<int> A_host(N);
  std::vector<int> B_host(N);
  std::vector<int> C_host(N);

  // fill the input vectors
  for(int i = 0; i < N; ++i){
    A_host[i] = rand()%100;
    B_host[i] = rand()%100;
  }

  // device memory
  int *A_device, *B_device, *C_device;

  hipMalloc(&A_device, bytes);
  hipMalloc(&B_device, bytes);
  hipMalloc(&C_device, bytes);

  // copying host to device memory
  hipMemcpy(A_device, A_host.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(B_device, B_host.data(), bytes, hipMemcpyHostToDevice);

  // device parameters
  int number_of_threads = 1 << 10;
  int number_of_blocks = (N + number_of_threads - 1)/number_of_threads;

  // kernel execution
  vector_addition<<<number_of_blocks, number_of_threads>>>(A_device, B_device, C_device, N);

  // copying device to host memory
  hipMemcpy(C_host.data(), C_device, bytes, hipMemcpyDeviceToHost);

  // verification
  check(A_host, B_host, C_host);

  // free device memory
  hipFree(A_device);
  hipFree(B_device);
  hipFree(C_device);

  std::cout << "The Vector Addition Worked!" << std::endl;

  return 0;
}



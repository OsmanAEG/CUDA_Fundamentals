
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

// parallel matrix multiplication
template<typename Scalar_type, typename Int_type>
__global__ void matrix_multiply(const Scalar_type* A, const Scalar_type* B,
                                Scalar_type* C, const Int_type N){
  // global index for the output matrix
  auto row = blockIdx.y * blockDim.y + threadIdx.y;
  auto col = blockIdx.x * blockDim.x + threadIdx.x;

  Scalar_type c_ij = 0.0;

  for(int k = 0; k < N; ++k){
    c_ij += A[row*N + k] * B[k*N + col];
  }

  C[row*N + col] = c_ij;
}

// check the matrix multiplication
template<typename Vector_type, typename Int_type>
void check_matrix_multiply(const Vector_type A, const Vector_type B,
                           const Vector_type C, const Int_type N){
  for(int i = 0; i < N; ++i){
    for(int j = 0; j < N; ++j){
      auto c_ij = 0.0;
      for(int k = 0; k < N; ++k){
        c_ij += A[i*N + k]*B[k*N + j];
      }
      // verifying answer
      assert(std::abs((C[i*N + j] - c_ij)/c_ij) < 1.0E-6);
    }
  }
}

int main(){
  // matrix dimensional values
  int N = 1 << 11;
  size_t bytes = N*N*sizeof(int);

  // input matrices on host memory
  std::vector<int> A_host(N*N);
  std::vector<int> B_host(N*N);

  // output matrix on host memory
  std::vector<int> C_host(N*N);

  // filling input matrices with random values
  std::generate(A_host.begin(), A_host.end(), [](){
    return rand()%100;
  });

  std::generate(B_host.begin(), B_host.end(), [](){
    return rand()%100;
  });

  // input matrices on device memory
  int *A_device, *B_device, *C_device;
  hipMalloc(&A_device, bytes);
  hipMalloc(&B_device, bytes);
  hipMalloc(&C_device, bytes);

  // copying host memory to device memory
  hipMemcpy(A_device, A_host.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(B_device, B_host.data(), bytes, hipMemcpyHostToDevice);

  // number of threads per block
  int number_of_threads = 32;

  // number of blocks per grid
  int number_of_blocks = N/number_of_threads;

  // setting up block and grid values
  dim3 threads(number_of_threads, number_of_threads);
  dim3 blocks(number_of_blocks, number_of_blocks);

  // kernel execution for matrix multiply
  matrix_multiply<<<blocks, threads>>>(A_device, B_device, C_device, N);

  // copying the output matrix from device to host memory
  hipMemcpy(C_host.data(), C_device, bytes, hipMemcpyDeviceToHost);

  // verifying answer
  check_matrix_multiply(A_host, B_host, C_host, N);

  // output the outcome
  std::cout << "Matrix Multiplication Results are Correct!" << std::endl;

  // freeing device memory
  hipFree(A_device);
  hipFree(B_device);
  hipFree(C_device);

  return 0;
}